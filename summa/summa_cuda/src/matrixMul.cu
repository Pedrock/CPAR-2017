#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 32

__global__ void matrixMulCUDA(float *A, float *B, float *C, int rowsA, int widthA, int rowsB, int widthB)
{
    const int block_x = blockIdx.x;
    const int block_y = blockIdx.y;
    const int thread_x = threadIdx.x;
    const int thread_y = threadIdx.y;

    const unsigned int row = block_y * BLOCK_SIZE + thread_y;
    const unsigned int col = block_x * BLOCK_SIZE + thread_x;

    float Cvalue = 0;

    float* Cs = C + widthB * BLOCK_SIZE * block_y + BLOCK_SIZE * block_x;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    const unsigned int numBLocks = (widthA + BLOCK_SIZE - 1) / BLOCK_SIZE;

    for (int m = 0; m < numBLocks; ++m)
    {
    	float* subA = A + widthA * BLOCK_SIZE * block_y + BLOCK_SIZE * m;
    	float* subB = B + widthB * BLOCK_SIZE * m + BLOCK_SIZE * block_x;

    	if (m * BLOCK_SIZE + thread_x < widthA && row < widthA)
    		As[thread_y][thread_x] = subA[thread_y * widthA + thread_x];
    	else
    		As[thread_y][thread_x] = 0;

    	if (m * BLOCK_SIZE + thread_y < rowsB && col < widthB)
    		Bs[thread_y][thread_x] = subB[thread_y * widthB + thread_x];
    	else
    		Bs[thread_y][thread_x] = 0;

        __syncthreads();

		#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
        	Cvalue += As[thread_y][k] * Bs[k][thread_x];
        }

        __syncthreads();
    }

    if (row < rowsA && col < widthB)
    	Cs[thread_y * widthB + thread_x] = Cvalue;
}


int matrixMultiply(int argc, char **argv, dim3 &dimsA, dim3 &dimsB)
{
	// Host
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int size_B = dimsB.x * dimsB.y;
    dim3 dimsC(dimsB.x, dimsA.y, 1);

    unsigned int mem_size_A = sizeof(float) * size_A;
    unsigned int mem_size_B = sizeof(float) * size_B;
    unsigned int mem_size_C = sizeof(float) * dimsC.y * dimsC.x;

    float *h_A = (float *)malloc(mem_size_A);
    float *h_B = (float *)malloc(mem_size_B);
    float *h_C = (float *)malloc(mem_size_C);

    for (int i = 0; i < size_A; i++) {
    	h_A[i] = 1.0;
    }

    for (int i = 0; i < dimsB.y; i++) {
    	for (int j = 0; j < dimsB.x; j++) {
    		h_B[i*dimsB.x + j] = i + 1.0;
    	}
    }

    // Device
    float *d_A, *d_B, *d_C;

    checkCudaErrors(hipMalloc((void **) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void **) &d_B, mem_size_B));
    checkCudaErrors(hipMalloc((void **) &d_C, mem_size_C));

    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Threads and grids
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((dimsC.x + threads.x - 1) / threads.x, (dimsC.y + threads.y - 1) / threads.y);

    // Start timer
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, NULL));

    // Multiply matrixes
    matrixMulCUDA<<< grid, threads >>>(d_A, d_B, d_C, dimsA.y, dimsA.x, dimsB.y, dimsB.x);

    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));

    // Get elapsed time
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Print time and performance
    double ops = 2.0 * (double)dimsA.y * (double)dimsA.x * (double)dimsB.x;
    double gigaFlops = (ops * 1.0e-9f) / (msecTotal / 1000.0f);
    printf("Performance: %.2f GFlop/s, Time: %.3f msec\n", gigaFlops, msecTotal);

    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    bool correct = true;
    double eps = 1.e-10;

    double expected_result = 0;
    for (int i = 0; i < dimsA.x; i++) expected_result += i + 1;

    for (int i = 0; i < (int)(dimsC.y * dimsC.x); i++)
    {
        double abs_err = fabs(h_C[i] - expected_result);
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], expected_result, eps);
            correct = false;
        }
    }
    printf("Result: %s\n", correct ? "success" : "failure");

    hipFree(d_A);
    free(h_A);
    hipFree(d_B);
    free(h_B);
    hipFree(d_C);
    free(h_C);

    return !correct;
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    if (argc != 4) {
        printf("Usage [height A] [width A = height B] [width B]\n");
        exit(0);
    }

    int devID = 0;
    checkCudaErrors(hipGetDevice(&devID));

    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int p = atoi(argv[3]);

    dim3 dimsA(n, m, 1); // A[m, n]
    dim3 dimsB(p, n, 1); // B[n, p]

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.y, dimsA.x, dimsB.y, dimsB.x);

    return matrixMultiply(argc, argv, dimsA, dimsB);
}
