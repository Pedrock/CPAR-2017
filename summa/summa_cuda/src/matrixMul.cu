#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 32

__global__ void matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = wA * BLOCK_SIZE * by;
    int aEnd   = aBegin + wA - 1;
    int aStep  = BLOCK_SIZE;
    int bBegin = BLOCK_SIZE * bx;
    int bStep  = BLOCK_SIZE * wB;

    float Csub = 0;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
    {
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        __syncthreads();

		#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}


int matrixMultiply(int argc, char **argv, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    for (int i = 0; i < size_A; i++)
    {
    	h_A[i] = 1.0;
    }

    for (int i = 0; i < dimsB.y; i++) {
    	for (int j = 0; j < dimsB.x; j++) {
    		h_B[i*dimsB.x + j] = i + 1.0;
    	}
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(1);
    }


    checkCudaErrors(hipMalloc((void **) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void **) &d_B, mem_size_B));
    checkCudaErrors(hipMalloc((void **) &d_C, mem_size_C));

    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    printf("Computing result using CUDA Kernel...\n");

    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, NULL));


    matrixMulCUDA<<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);


    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecTotal / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecTotal,
        flopsPerMatrixMul,
        threads.x * threads.y);

    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));


    printf("Checking computed result for correctness: ");
    bool correct = true;
    double eps = 1.e-10 ; // machine zero

    double expected_result = 0;
    for (int i = 0; i < dimsB.y; i++)
    	expected_result += i + 1;

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - expected_result);
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], expected_result, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");



    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return !correct;
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    dim3 dimsA(4*BLOCK_SIZE, 4*BLOCK_SIZE, 1);
    dim3 dimsB(4*BLOCK_SIZE, 4*BLOCK_SIZE, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA"))
    {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA"))
    {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB"))
    {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB"))
    {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n", dimsA.x, dimsB.y);
        exit(1);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    int matrix_result = matrixMultiply(argc, argv, dimsA, dimsB);

    exit(matrix_result);
}
